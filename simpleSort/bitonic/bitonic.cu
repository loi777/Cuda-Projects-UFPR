#include <cstdio>
#include <cstdlib>
#include <hip/hip_runtime.h>
#include <iostream>
#include <limits>

#define BLOCKS 1
#define THREADS 1024

__device__ inline void swap(int &a, int &b) {
  int tmp = a;
  a = b;
  b = tmp;
}


__global__ void bitonicSort(int *values, int size) {
  extern __shared__ int shared[];

  // Get thread index within the block
  const int tid = threadIdx.x + blockIdx.x * blockDim.x;

  // Ensure we do not read out of bounds
  if (tid < size) {
    // Copy input to shared memory
    shared[threadIdx.x] = values[tid];
    __syncthreads();

    // Perform Bitonic sort
    for (int k = 2; k <= size; k *= 2) {
      // Bitonic merge
      for (int j = k / 2; j > 0; j /= 2) {
        int ixj = threadIdx.x ^ j;

        // Ensure ixj is within bounds
        if (ixj > threadIdx.x && ixj < blockDim.x) {
          if ((threadIdx.x & k) == 0) {
            if (shared[threadIdx.x] > shared[ixj]) { swap(shared[threadIdx.x], shared[ixj]); }
          } else {
            if (shared[threadIdx.x] < shared[ixj]) { swap(shared[threadIdx.x], shared[ixj]); }
          }
        }
        __syncthreads();
      }
    }

    // Write the result back to global memory
    values[tid] = shared[threadIdx.x];
  }
}


// Function to get the next power of two greater than or equal to the given value
unsigned int getNextPowerOfTwo(size_t value) {
  unsigned int power = 1;
  while (power < value) { power <<= 1; }

  return power;
}

// Function to generate the padded array
unsigned int* generatePaddedArray(size_t realSize, size_t nextPowerOfTwo) {
  // Ensure nextPowerOfTwo is a power of two
  if ((nextPowerOfTwo & (nextPowerOfTwo - 1)) != 0) {
    std::cerr << "nextPowerOfTwo must be a power of two." << std::endl;
    return nullptr;
  }

  // Allocate memory for the array
  unsigned int *array = new unsigned int[nextPowerOfTwo];
  // Fill the array with random values for the first part
  for (size_t i = 0; i < realSize; ++i) { array[i] = std::rand(); }
  // Fill the rest of the array with the maximum unsigned int value
  for (size_t i = realSize; i < nextPowerOfTwo; ++i) { array[i] = UINT_MAX; }

  return array;
}



// Problemas a resolver:
// 1 - Maximo de threads por bloco: 1024
// 2 - Tamanho maximo da shared memory: 49152 -> Tamanho maximo de uint: 1536
int main(int argc, char** argv) {
  unsigned int realSize = 1024;
  unsigned int powerSize = getNextPowerOfTwo(realSize);
  unsigned int* array = generatePaddedArray(realSize, powerSize);

  //std::cout << "Input: " << powerSize << std::endl;
  //for (int i=0; i<powerSize ;i++)
  //  std::cout << array[i] << " ";
  //std::cout << std::endl;

  int *d_values;
  hipMalloc((void**)&d_values, sizeof(int) * powerSize);
  hipMemcpy(d_values, array, sizeof(int) * powerSize, hipMemcpyHostToDevice);

  bitonicSort<<<BLOCKS, THREADS, THREADS * sizeof(unsigned int)>>>(d_values, powerSize);

  hipMemcpy(array, d_values, sizeof(int) * powerSize, hipMemcpyDeviceToHost);
  hipFree(d_values);

  //std::cout << "Output: " << std::endl;
  //for (int i=0; i<powerSize ;i++)
  //  std::cout << array[i] << " ";
  //std::cout << std::endl;

  bool passed = true;
  for(int i = 1; i < realSize; i++)
    if (array[i-1] > array[i]) {
      passed = false;
      printf("Falha na posicao [%d] com [%d]\n", i-1, i);
    }

  printf("Test %s\n", passed ? "PASSED" : "FAILED");
}
