#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdlib>
#include <ctime>
#include <algorithm>
#include <hip/hip_runtime.h>
#include <thrust/sort.h>
#include <thrust/device_vector.h>

#include <hip/hip_cooperative_groups.h>

#include "simpleSort.cuh"
#include "histogramM.cuh"
#include "bitonicM.cuh"

namespace cg = cooperative_groups;

#include "chrono.c"

typedef unsigned int u_int;



//---------------------------------------------------------------------------------



__global__ void verifySort(u_int* d_arr1, u_int* d_arr2, u_int size) {
  u_int posi = (blockDim.x*blockIdx.x) + threadIdx.x;

  while(posi < size) {
    if (d_arr1[posi] != d_arr2[posi]) {
      //printf("FALHA NA VERIFICACAO!!!\n     valores encontrados: [%d]%d [%d]%d\n", posi, d_arr1[posi], posi, d_arr2[posi]);
    }

    posi += blockDim.x;
  }
}


void verifySortProxy(u_int* h_arr1, u_int* h_arr2, u_int size) {
  u_int *d_arr1, *d_arr2;
  hipMalloc((void**)&d_arr1, sizeof(u_int) * size);
  hipMalloc((void**)&d_arr2, sizeof(u_int) * size);

  //--
  verifySort<<<1, THREADS >>>(d_arr1, d_arr2, size);
  //--

  hipFree(d_arr1);
  hipFree(d_arr2);
}



//---------------------------------------------------------------------------------


// Create and generate a random array of nElements
// returns as a pointer
u_int* genRandomArray(int nElem) {
  u_int* array = new u_int[nElem];

  for (int i = 0; i < nElem; ++i) {
    int a = std::rand() % 50;
    int b = std::rand();
    u_int v = a * 100 + b;
    array[i] = v;
  }

  return array;
}


//---------------------------------------------------------------------------------


u_int check_parameters(int argc){
  if (argc != 4) {
    std::cerr << "Usage: ./simpleSort <nTotalElements> <h> <nR>" << std::endl;
    return EXIT_FAILURE;
  }
  return 0;
}


//---------------------------------------------------------------------------------


void cudaResetVariables(u_int *HH, u_int *Hg, u_int *SHg, u_int *PSv, u_int h){
  hipMemset(HH,  0, NB * h * sizeof(u_int));
  hipMemset(PSv, 0, NB * h * sizeof(u_int));
  hipMemset(Hg,  0, h * sizeof(u_int));
  hipMemset(SHg, 0, h * sizeof(u_int));
}


//---------------------------------------------------------------------------------



// CPU level recursion function that uses histogram to constantly reduce the size of the array
// when this goes below the shared memory limit then bitonic sort is used to sort the array.
void recursionBitonic(u_int* d_array, u_int p_start, u_int p_end, u_int histograms) {
  u_int a_size = (p_end-p_start);                             // obtem o tamanho em elementos dessa particao
  u_int h_min = UINT32_MAX;
  u_int h_max = 0;
  H_getDeviceMinMax(d_array, p_start, a_size, &h_min, &h_max);

  u_int binWidth = H_getBinSize(h_min, h_max, histograms);      // obtem as ranges dos conjuntos numericos/bins
  u_int SEG_SIZE = (ceil((float)a_size/((float)NB)));         // obtem o tamanho em elementos 

  //--

  if (a_size < POW2LIMIT) {    // esse segmento eh pequeno o suficiente, ordena com bitonic

    B_bitonicProxy(&d_array[p_start], a_size);

  } else {      // esse segmento eh mt grande, particiona com histogramas

    u_int *d_partitioned, *d_HH, *d_Hg, *d_horizontalS, *d_verticalS;
    u_int h_horizontalS[histograms];
    hipMalloc((void**)&d_partitioned,  a_size                  * sizeof(u_int));  // device output partitionec data
    hipMalloc((void**)&d_HH,           NB * histograms         * sizeof(u_int));  // device histogram matrix
    hipMalloc((void**)&d_Hg,           histograms              * sizeof(u_int));  // device histogram sum
    hipMalloc((void**)&d_horizontalS,  histograms              * sizeof(u_int));  // device histogram prefix sum
    hipMalloc((void**)&d_verticalS,    NB * histograms         * sizeof(u_int));  // device matrix vertical prefix sum

    ////==== ALOCA MEMORIA CUDA

    
    cudaResetVariables(d_HH, d_Hg, d_horizontalS, d_verticalS, histograms);
    H_getHistogram        <<<NB, THREADS, histograms*sizeof(u_int)>>>(d_HH, d_Hg, histograms, &d_array[p_start], a_size, h_min, h_max, SEG_SIZE, binWidth);
    H_horizontalScan      <<<1,  THREADS, histograms*sizeof(u_int)>>>(d_Hg, d_horizontalS, histograms);
    H_verticalScan        <<<NB, THREADS, histograms*sizeof(u_int)>>>(d_HH, d_verticalS, histograms);
    H_Partitioner         <<<NB, THREADS, histograms*sizeof(u_int)>>>(d_HH, d_horizontalS, d_verticalS, histograms, &d_array[p_start], d_partitioned, a_size, h_min, h_max, SEG_SIZE, binWidth);
    hipMemcpy(h_horizontalS, d_horizontalS, histograms * sizeof(u_int), hipMemcpyDeviceToHost); // salva no host o histograma horizontal para saber a posicao das part.
 

    ////==== PARTICIONA USANDO HISTOGRAMA

    for (int p_hist = 1; p_hist < histograms; p_hist++) {
      recursionBitonic(d_array, h_horizontalS[p_hist-1], h_horizontalS[p_hist], histograms);
    }
    recursionBitonic(d_array, h_horizontalS[histograms-1], p_end, histograms);  // o ultimo ponto quebra a logica do loop e eh feito fora

    ////==== CONTINUA A RECURSAO

    hipFree(d_partitioned);
    hipFree(d_HH);
    hipFree(d_Hg);
    hipFree(d_horizontalS);
    hipFree(d_verticalS);

    ////==== LIMPA MEMORIA CUDA
  }
}



//---------------------------------------------------------------------------------



int main(int argc, char* argv[]) {
  if (check_parameters(argc)) { return EXIT_FAILURE; }
  std::srand(std::time(nullptr));

  u_int nTotalElements = std::stoi(argv[1]);                    // Numero de elementos
  u_int h = std::stoi(argv[2]);                                 // Numero de histogramas/recursao
  u_int nR = std::stoi(argv[3]);                                // Numero de chamadas do kernel
  u_int *h_Input = genRandomArray(nTotalElements);              // Vetor de entrada
  u_int *h_Output_bi = new u_int[nTotalElements];                  // Vetor final BITONIC
  u_int *h_Output_th = new u_int[nTotalElements];                  // Vetor final THRUST

  ////====  GET GLOBAL VARIABLES

  u_int *d_input;
  hipMalloc((void**)&d_input, nTotalElements * sizeof(u_int));   // device input data

  ////====  GET CUDA MEMORY

  chronometer_t chrono_Thrust, chrono_Hist;
  chrono_reset(&chrono_Thrust);
  chrono_reset(&chrono_Hist);

  ////====  GET CHRONO VARIABLES

  // Information printing, a pedido do Zola
  // variaveis apenas usadas nesse print
  u_int nMin = *std::min_element(h_Input, h_Input + nTotalElements);  // obtem o min dessa particao
  u_int nMax = *std::max_element(h_Input, h_Input + nTotalElements);  // obtem o max dessa particao
  u_int binWidth = H_getBinSize(nMin, nMax, h);                   // obtem as ranges dos conjuntos numericos/bins

  std::cout << "Min: " << nMin << " | Max: " << nMax << std::endl;
  std::cout << "Largura da Faixa: " << binWidth << std::endl;

  ////====  PRINT DE INFORMAÇÃO

  hipMemcpy(d_input, h_Input, nTotalElements * sizeof(u_int), hipMemcpyHostToDevice);     // get input array to device
  
  chrono_start(&chrono_Hist);
  recursionBitonic(d_input, 0, nTotalElements, h);                                          // Begin bitonic sort
  chrono_stop(&chrono_Hist);

  hipMemcpy(h_Output_bi, d_input, nTotalElements * sizeof(u_int), hipMemcpyDeviceToHost); // get output array to host

  ////====  BITONIC RECURSION

  std::cerr << "DEBUGG  1\n";

  thrust::device_vector<u_int> d_vec(h_Input, h_Input + nTotalElements);  // get input array to thrust

  std::cerr << "DEBUGG  1.2\n";

  chrono_start(&chrono_Thrust);
  thrust::sort(d_vec.begin(), d_vec.end());                               // Begin thrust sort
  chrono_stop(&chrono_Thrust);

  std::cerr << "DEBUGG  1.3\n";

  thrust::copy(d_vec.begin(), d_vec.end(), h_Output_th);                  // get output array to host

  std::cerr << "DEBUGG  2\n";

  ////====  THRUST SORT

  verifySortProxy(h_Output_th, h_Output_bi, nTotalElements);

  ////====  VERIFICA SORT

  printf("\n----THRUST\n");
  printf("Delta time: " );
  chrono_report_TimeInLoop( &chrono_Thrust, (char *)"thrust sort", nR);

  double thrust_time_seconds = (double) chrono_gettotal( &chrono_Thrust )/((double)1000*1000*1000);
  printf( "Tempo em segundos: %lf s\n", thrust_time_seconds );
  printf( "Vazão: %lf INT/s\n", (nTotalElements)/thrust_time_seconds );
  
  //--

  printf("\n----HISTOGRAM\n");
  printf("Delta time: " );
  chrono_report_TimeInLoop( &chrono_Hist, (char *)"histogram sort", nR);

  double reduce_time_seconds = (double) chrono_gettotal( &chrono_Hist )/((double)1000*1000*1000);
  printf( "Tempo em segundos: %lf s\n", reduce_time_seconds );
  printf( "Vazão: %lf INT/s\n", (nTotalElements)/reduce_time_seconds );

  printf("\n--Tempo em relacao ao Thrust\n");
  printf("Em segundos: %lf\n", reduce_time_seconds - thrust_time_seconds);
  printf("Em porcento: %d\n", (int)((thrust_time_seconds/reduce_time_seconds)*100.0));

  ////==== PRINT RESULTADOS

  hipFree(d_input);

  ////==== FREE MEMORY

  return EXIT_SUCCESS;
}

