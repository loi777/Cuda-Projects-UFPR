#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdlib>
#include <ctime>
#include <algorithm>
#include <hip/hip_runtime.h>
#include <thrust/sort.h>
#include <thrust/device_vector.h>

#include <hip/hip_cooperative_groups.h>

namespace cg = cooperative_groups;

#include "chrono.c"

typedef unsigned int u_int;

#define NP 2             // Number of processors
#define BLOCKS 28         // Number of blocks per processor
#define NB NP*BLOCKS     // Total number of blocks
#define THREADS 1024     // Number of threads per block

#define BINFIND(min, max, val, binSize, binQtd) (val >= max ? binQtd-1 : (val - min) / binSize)

//--------------------------------------------------------------------------

// returns the size of the number group of each bin
// needs some strange calculations due to precision error
u_int getBinSize(u_int min, u_int max, int segCount) {
  u_int binSize = max - min;
  if ((binSize % segCount) == 0) {
    // complete division
    binSize /= segCount;
  } else {
    // incomplete division
    binSize /= segCount;
    binSize++;
  }

  return binSize;
}


// Kernel para calcular histogramas em particoes
// Cada bloco eh responsavel por um histograma (linha da matriz)
__global__ void blockAndGlobalHisto(u_int *HH, u_int *Hg, u_int h, u_int *Input, u_int nElements, u_int nMin, u_int nMax, u_int segSize, u_int binWidth) {
    // Alloca shared memory para UM histograma
    extern __shared__ int _HH[];
    if (threadIdx.x < h) { _HH[threadIdx.x] = 0; }
    __syncthreads();

    //---

    // Inicio da particao no vetor
    int blcStart = (blockIdx.x * segSize);    // bloco positionado na frente daquele que veio anterior a ele
    int thrdPosi = threadIdx.x;              // 1 elemento por thread, starts as exactly the thread.x

    while(thrdPosi < segSize && ((blcStart+thrdPosi) < nElements)) {
        // Loop enquanto a thread estiver resolvendo elementos validos dentro do bloco e do array
        u_int val = Input[blcStart + thrdPosi];    // get value
        int posi = BINFIND(nMin, nMax, val, binWidth, h);
        atomicAdd(&_HH[posi], 1);  // add to its corresponding segment
        atomicAdd(&Hg[posi], 1);  // add to its corresponding segment

        thrdPosi += blockDim.x; // thread pula para frente, garantindo que nao ira processar um valor ja processado
    }

    __syncthreads();

    //--

    // Passa os resultados da shared memory para matriz
    // deixar isso a cargo da thread 0 eh mais modular que mandar uma pra uma
    if (threadIdx.x < h)
      atomicAdd(&HH[(blockIdx.x * h) + threadIdx.x], _HH[threadIdx.x]);
    __syncthreads();
}


// calculates the scan of the global histogram and saves it into the horizontal scan
__global__ void globalHistoScan(u_int *Hg, u_int *SHg, u_int h){
    // Obtem shared memory para o histogram horizontal
    extern __shared__ u_int _SHg[];
    if (threadIdx.x < h) { _SHg[threadIdx.x] = 0; }
    __syncthreads();

    //--

    u_int thrdPosi = threadIdx.x;         // starts as thread ID

    //--

    while (thrdPosi < h) {
      // Loop while inside the histogram
      u_int sum = 0;
      for (int i = thrdPosi-1; i >= 0; i--) {
        sum += Hg[i]; // makes the individual sum of every index before this one
      }
      _SHg[thrdPosi] = sum;

      //--

      thrdPosi += blockDim.x; // go to the next element
    }

    __syncthreads();

    //--

    // Passa os resultados da shared memory para o scan
    // deixar isso a cargo da thread 0 eh mais modular que mandar uma pra uma
    if (threadIdx.x < h)
      SHg[threadIdx.x] = _SHg[threadIdx.x];
    __syncthreads();
}


// calculates the scan of each non-global histogram, saving it in different lines of the vertical scan
__global__ void verticalScanHH(u_int *HH, u_int *PSv, u_int h){
    // Obtem shared memory para o histogram horizontal
    extern __shared__ u_int _PSv[];
    if (threadIdx.x < h) { _PSv[threadIdx.x] = 0; }
    __syncthreads();

    //--

    u_int thrdPosi = threadIdx.x;     // Thread por coluna

    //--

    while (thrdPosi < h) {
      int sum = 0;
      for (int i=blockIdx.x-1; i>=0; i--){
        sum += HH[i*h + thrdPosi];
      }
      _PSv[thrdPosi] = sum;

      thrdPosi += blockDim.x; // go to the next element
    }
    __syncthreads();

    //--

    // Passa os resultados da shared memory para o scan
    // deixar isso a cargo da thread 0 eh mais modular que mandar uma pra uma
    if (threadIdx.x < h)
      PSv[blockIdx.x*h + threadIdx.x] = _PSv[threadIdx.x];
    __syncthreads();
}


// Uses the consultation table to separate the groups of numbers according to their bins
// saves in output device memory
__global__ void PartitionKernel(u_int *HH, u_int *SHg, u_int *PSv, u_int h, u_int *Input, u_int *Output, u_int nElements, u_int nMin, u_int nMax, u_int segSize, u_int binWidth) {
    extern __shared__ u_int _HLsh[];
    if (threadIdx.x < h) { _HLsh[threadIdx.x] = 0; }
    __syncthreads();

    // Thread ID and total threads
    u_int thrdPosi = threadIdx.x; 
    u_int totalThreads = blockDim.x;

    // Calculate the indices for shared memory
    while (thrdPosi < h) {
        _HLsh[thrdPosi] = SHg[thrdPosi] + PSv[blockIdx.x * h + thrdPosi];
        thrdPosi += totalThreads;
    }
    __syncthreads();

    // Reset thread position for the next phase
    thrdPosi = threadIdx.x;

    // Process elements in the segment
    while (thrdPosi < segSize && ((blockIdx.x * segSize + thrdPosi) < nElements)) {
        u_int val = Input[blockIdx.x * segSize + thrdPosi]; 
        u_int posi = BINFIND(nMin, nMax, val, binWidth, h);

        // Atomic operation to update the output array
        u_int index = atomicAdd(&_HLsh[posi], 1);    // Get the current position and increment it atomically
        if (index < nElements)
          Output[index] = val;                         // Write the value to the output array

        thrdPosi += totalThreads;
    }
    __syncthreads();
}


void thrustSortProxy(u_int* h_array, u_int start, u_int end) {
  thrust::device_vector<u_int> d_vec(&h_array[start], &h_array[start] + (end-start));

  thrust::sort(d_vec.begin(), d_vec.end());

  thrust::copy(d_vec.begin(), d_vec.end(), &h_array[start]);
}


//---------------------------------------------------------------------------------


//hipMemcpy(Output, d_Output, nTotalElements * sizeof(u_int), hipMemcpyDeviceToHost);
//verifySort(Input, Output, nTotalElements);
void verifySort(u_int *Input, u_int *Output, u_int nElements, chronometer_t *chrono_Thrust, u_int k) {
  thrust::device_vector<u_int> th_Input(Input, Input + nElements);
  thrust::device_vector<u_int> th_Output(Output, Output + nElements);
  
  chrono_start(chrono_Thrust);
  thrust::sort(th_Input.begin(), th_Input.end());
  chrono_stop(chrono_Thrust);

  bool isSorted = thrust::equal(th_Input.begin(), th_Input.end(), th_Output.begin());
  if (isSorted) { std::cout << "Sort " << k << " verification: SUCCESS" << std::endl; } 
  else          { std::cout << "Sort " << k << " verification: FAILURE" << std::endl; }
}


//---------------------------------------------------------------------------------


// Create and generate a random array of nElements
// returns as a pointer
u_int* genRandomArray(int nElem) {
  u_int* array = new u_int[nElem];

  for (int i = 0; i < nElem; ++i) {
    int a = std::rand() % 50;
    int b = std::rand();
    u_int v = a * 100 + b;
    array[i] = v;
  }

  return array;
}


//---------------------------------------------------------------------------------


u_int check_parameters(int argc){
  if (argc != 4) {
    std::cerr << "Usage: ./simpleSort <nTotalElements> <h> <nR>" << std::endl;
    return EXIT_FAILURE;
  }
  return 0;
}


//---------------------------------------------------------------------------------

void cudaResetVariables(u_int *HH, u_int *Hg, u_int *SHg, u_int *PSv, u_int h){
  hipMemset(HH,  0, NB * h * sizeof(u_int));
  hipMemset(PSv, 0, NB * h * sizeof(u_int));
  hipMemset(Hg,  0, h * sizeof(u_int));
  hipMemset(SHg, 0, h * sizeof(u_int));
}


//---------------------------------------------------------------------------------

int main(int argc, char* argv[]) {
  if (check_parameters(argc)) { return EXIT_FAILURE; }
  std::srand(std::time(nullptr));

  u_int nTotalElements = std::stoi(argv[1]);                    // Numero de elementos
  u_int h = std::stoi(argv[2]);                                 // Numero de histogramas
  u_int nR = std::stoi(argv[3]);                                // Numero de chamadas do kernel
  u_int *Input = genRandomArray(nTotalElements);                // Vetor de entrada
  //u_int nTotalElements = 18;
  //u_int h = 6;
  //u_int nR = 4;
  //u_int Input[] = {2, 4, 33, 27, 8, 10, 42, 3, 12, 21, 10, 12, 15, 27, 38, 45, 18, 22};
  u_int *Output = new u_int[nTotalElements];                      // Vetor ordenado
  u_int *h_SHg = new u_int[h];                      // Vetor ordenado
  u_int SEG_SIZE = (ceil((float)nTotalElements/((float)NB)));
  chronometer_t chrono_Thrust, chrono_Hist;

  // Busca menor valor, maior valor e o comprimento do bin
  u_int nMin = *std::min_element(Input, Input + nTotalElements);
  u_int nMax = *std::max_element(Input, Input + nTotalElements);
  u_int binWidth = getBinSize(nMin, nMax, h);

  // Information printing, a pedido do Zola
  std::cout << "Min: " << nMin << " | Max: " << nMax << std::endl;
  std::cout << "Largura da Faixa: " << binWidth << std::endl;

  // Aloca cores e copia para GPU
  u_int *d_Input, *d_Output, *HH, *Hg, *SHg, *PSv;
  hipMalloc((void**)&d_Input,  nTotalElements * sizeof(u_int));  // device input data
  hipMalloc((void**)&d_Output, nTotalElements * sizeof(u_int));  // device input sorted data
  hipMalloc((void**)&HH,       NB * h         * sizeof(u_int));  // device histogram matrix
  hipMalloc((void**)&Hg,       h              * sizeof(u_int));  // device histogram sum
  hipMalloc((void**)&SHg,      h              * sizeof(u_int));  // device histogram prefix sum
  hipMalloc((void**)&PSv,      NB * h         * sizeof(u_int));  // device matrix vertical prefix sum
  hipMemcpy(d_Input, Input, nTotalElements * sizeof(u_int), hipMemcpyHostToDevice);

  chrono_reset(&chrono_Thrust);
  chrono_reset(&chrono_Hist);

  for (int k = 0; k < nR; k++) {
    cudaResetVariables(HH, Hg, SHg, PSv, h);
    chrono_start(&chrono_Hist);
    blockAndGlobalHisto<<<NB, THREADS, h*sizeof(u_int)>>>(HH, Hg, h, d_Input, nTotalElements, nMin, nMax, SEG_SIZE, binWidth);
    globalHistoScan    <<<1,  THREADS, h*sizeof(u_int)>>>(Hg, SHg, h);
    verticalScanHH     <<<NB, THREADS, h*sizeof(u_int)>>>(HH, PSv, h);
    PartitionKernel    <<<NB, THREADS, h*sizeof(u_int)>>>(HH, SHg, PSv, h, d_Input, d_Output, nTotalElements, nMin, nMax, SEG_SIZE, binWidth);
    // launch kernel that that sorts the inside of each bin partition
    hipMemcpy(h_SHg, SHg, h * sizeof(u_int), hipMemcpyDeviceToHost);
    hipMemcpy(Output, d_Output, nTotalElements * sizeof(u_int), hipMemcpyDeviceToHost);
    u_int start, end = 0;
    for (u_int bin = 1; bin < h; bin++) {
      // call a bitonic sort for every bin
      start = end;
      end = h_SHg[bin];
      thrustSortProxy(Output, start, end);
    }
    start = end;
    end = nTotalElements;
    thrustSortProxy(Output, start, end);
    chrono_stop(&chrono_Hist);

    verifySort(Input, Output, nTotalElements, &chrono_Thrust, k);
  }

  // ---

  printf("\n----THRUST\n");
  printf("Delta time: " );
  chrono_report_TimeInLoop( &chrono_Thrust, (char *)"thrust sort", nR);

  double thrust_time_seconds = (double) chrono_gettotal( &chrono_Thrust )/((double)1000*1000*1000);
  printf( "Tempo em segundos: %lf s\n", thrust_time_seconds );
  printf( "Vazão: %lf INT/s\n", (nTotalElements)/thrust_time_seconds );
  
  //--

  printf("\n----HISTOGRAM\n");
  printf("Delta time: " );
  chrono_report_TimeInLoop( &chrono_Hist, (char *)"histogram sort", nR);

  double reduce_time_seconds = (double) chrono_gettotal( &chrono_Hist )/((double)1000*1000*1000);
  printf( "Tempo em segundos: %lf s\n", reduce_time_seconds );
  printf( "Vazão: %lf INT/s\n", (nTotalElements)/reduce_time_seconds );

  printf("\n--Tempo em relacao ao Thrust\n");
  printf("Em segundos: %lf\n", reduce_time_seconds - thrust_time_seconds);
  printf("Em porcento: %d\n", (int)((thrust_time_seconds/reduce_time_seconds)*100.0));

  //--

  hipFree(d_Input);
  hipFree(d_Output);
  hipFree(HH);
  hipFree(Hg);
  hipFree(SHg);
  hipFree(PSv);

  //delete[] Input;
  delete[] Output;

  return EXIT_SUCCESS;
}

