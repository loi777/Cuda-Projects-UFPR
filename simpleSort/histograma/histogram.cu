#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <limits>

#include <thrust/sort.h>            // THRUST LIB
#include <thrust/device_vector.h>   // THRUST LIB

typedef unsigned int u_int;

#define BLOCKS 8                            // one block for one histogram
#define THREADS 12                                 // n of threads

#define ARRAYSIZE 18                        // Size of the input array
#define HIST_SEGMENTATIONS 6                // BINS number

#define SEG_SIZE (ceil((float)ARRAYSIZE/(float)BLOCKS))   // Every block will solve this size, minimun of 1

#define HISTOGRAM (BLOCKS*HIST_SEGMENTATIONS)             // the full histogram, block:y | segmentation:x

#define BINSTART(min, binSize, i) ((u_int)((binWidth*i)+min))
#define BINEND(min, binSize, i) ((u_int)(BINSTART(min, binSize, (i+1))-1))
#define BINFIND(min, max, val, binSize, binQtd) (val >= max ? binQtd-1 : (val - min) / binSize)



//---------------------------------------------------------------



// Create and generate a random array of nElements
// returns as a pointer
u_int* genRandomArray(int nElem) {
  u_int* array = new u_int[nElem];

  for (int i = 0; i < nElem; ++i) {
    int a = std::rand() % 50;
    int b = std::rand();
    u_int v = a * 100 + b;
    array[i] = v;
  }

  return array;
}



//---------------------------------------------------------------



//GPU Kernel Implementation of Bitonic Sort
__global__ void bitonicSortGPU(u_int* arr, int j, int k, u_int start, u_int end) {
    unsigned int i, ij;

    i = threadIdx.x + blockDim.x * blockIdx.x;

    ij = i ^ j;

    if (i < end-start && ij < end-start) {

      printf("B(%d)T(%d)S(%d)E(%d)  ==  comp [%d]%d < [%d]%d\n", blockIdx.x, threadIdx.x, start, end, i, arr[i+start], ij, arr[ij+start]);
      if (ij > i) {                   // ij is to the right of i
        if ((i & k) == 0) {           // if the thread is going forward or back
          if (arr[i+start] > arr[ij+start]) {     // only invert  
            printf("B(%d)T(%d)  ==  inverting [%d]%d <-> [%d]%d\n", blockIdx.x, threadIdx.x, i, arr[i+start], ij, arr[ij+start]);

            int temp = arr[i+start];  // arr[i] receives arr[ij]
            arr[i+start] = arr[ij+start];
            arr[ij+start] = temp;
          }
        } else {
          if (arr[i+start] < arr[ij+start]) {
            printf("B(%d)T(%d)  ==  inverting [%d]%d <-> [%d]%d\n", blockIdx.x, threadIdx.x, i, arr[i+start], ij, arr[ij+start]);

            int temp = arr[i+start];  // arr[i] receives arr[ij]
            arr[i+start] = arr[ij+start];
            arr[ij+start] = temp;
          }
        }
      }

    }

    __syncthreads();
}


// Um proxy para a chama do bitonic
void bitonicSortProxy(u_int* d_array, u_int start, u_int end) {
  int k = 2;  // nao precisa ordernar grupos de 1
  while (k <= end-start) {

    for (int j = k >> 1; j > 0; j = j >> 1) {
        std::cout << "Bitonic Size[" << end-start << "] Start[" << start << "] batch[" << k << "] segments [" << j << "]\n";

        bitonicSortGPU<<<BLOCKS, THREADS>>>(d_array, j, k, start, end);
    }

    k <<= 1;
  }
}



//---------------------------------------------------------------



// 0 for wrong | 1 for correct
int verifySort(u_int* arr, u_int size) {
  for (int i = 1; i < size; i++) {
    if (arr[i-1] > arr[i]) {
      printf("Array wrong at: [%d]%d == [%d]%d\n", i-1, arr[i-1], i, arr[i]);
      return 0;
    }
  }
  
  //--

  return 1;
}



//---------------------------------------------------------------



// returns the size of the number group of each bin
// needs some strange calculations due to precision error
u_int getBinSize(u_int min, u_int max, int segCount) {
  u_int binSize = max - min;
  if ((binSize % segCount) == 0) {
    // complete division
    binSize /= segCount;

  } else {
    // incomplete division
    binSize /= segCount;
    binSize++;

  }

  return binSize;
}


// returns the min value of an Array
u_int getMin(u_int* Array, int nElem) {
  u_int min = UINT_MAX;

  for (int i = 0; i < nElem; ++i) {
    if (Array[i] < min) {
        min = Array[i];
    }
  }

  return min;
}


// returns the max value of an Array
u_int getMax(u_int* Array, int nElem) {
  u_int max = 0;    // minimun value of an unsigned variable is 0

  for (int i = 0; i < nElem; ++i) {
    if (Array[i] > max) {
        max = Array[i];
    }
  }

  return max;
}



//---------------------------------------------------------------



// FOR INTERNAL AND DEBUGG USE
// Debugg function to print an array
void intPrintArray(int* a, int size) {
    // Print the generated array, do not allow this with arrays of billions
    for (int i = 0; i < size; i++) {
        std::cout << a[i] << " ";
    }
    std::cout << std::endl;
}


// FOR INTERNAL AND DEBUGG USE
// Debugg function to print an array
void uintPrintArray(u_int* a, int size) {
    // Print the generated array, do not allow this with arrays of billions
    for (int i = 0; i < size; i++) {
        std::cout << a[i] << " ";
    }
    std::cout << std::endl;
}


// FOR INTERNAL AND DEBUGG USE
// Debugg function to print an array
void printSegmentations(u_int min, u_int max, u_int* a, int size, int segCount) {
    u_int binWidth = getBinSize(min, max, segCount);  

    //--

    std::cout << "min: " << min << " | max: " << max << std::endl;
    std::cout << "Bin Size: " << binWidth << std::endl;
    for (int i = 0; i < segCount; i++) {
        std::cout << "Seg|Bin [" << i << "]: " << BINSTART(min, binWidth, i) << " to " << BINEND(min, binWidth, i) << "\n";
    }
    std::cout << std::endl;
}



//---------------------------------------------------------------



// Kernel para calcular histogramas em particoes
// Cada bloco eh responsavel por um histograma (linha da matriz)
__global__ void calculateHistogram(const u_int *input, int *histograms, int *histogram_T, int arraySize, int segSize, int segCount, u_int minVal, u_int maxVal, u_int binWidth) {
    // Alloca shared memory para UM histograma
    extern __shared__ int sharedHist[];

    //---

    // Inicio da particao no vetor
    int blcStart = (blockIdx.x * segSize);    // bloco positionado na frente daquele que veio anterior a ele
    int thrPosi = threadIdx.x;              // 1 elemento por thread, starts as exactly the thread.x

    while(thrPosi < segSize && ((blcStart+thrPosi) < arraySize)) {
        // Loop enquanto a thread estiver resolvendo elementos validos dentro do bloco e do array

        u_int val = input[blcStart + thrPosi];    // get value
        int posi = BINFIND(minVal, maxVal, val, binWidth, segCount);
        atomicAdd(&sharedHist[posi], 1);  // add to its corresponding segment
        atomicAdd(&histogram_T[posi], 1);  // add to its corresponding segment

        thrPosi += blockDim.x; // thread pula para frente, garantindo que nao ira processar um valor ja processado
        // saira do bloco quando terminar todos os pixeis dos quais eh responsavel
    }

    __syncthreads();

    //--

    // Passa os resultados da shared memory para matriz
    // deixar isso a cargo da thread 0 eh mais modular que mandar uma pra uma
    if (threadIdx.x == 0) {
      for (int i = 0; i < segCount; i++) {
        atomicAdd(&histograms[(blockIdx.x * segCount) + i], sharedHist[i]); 
      }
    }
    // Y: (blockIdx.x * segCount)
    // X: threadIdx.x

    __syncthreads();
}



//---------------------------------------------------------------



// calculates the scan of the global histogram and saves it into the horizontal scan
__global__ void calculateHorizontalScan(int *histogram_T, int *scan, int segCount) {
    int threadPosi = threadIdx.x;         // starts as thread ID

    //--

    while(threadPosi < segCount) {
      // Loop while inside the histogram

      int sum = 0;

      for (int i = threadPosi-1; i >= 0; i--) {
        // makes the individual sum of every index before this one
        sum += histogram_T[i];
      }

      scan[threadPosi] = sum;

      //--

      threadPosi += blockDim.x; // go to the next element
    }

    //--

    __syncthreads();
}


// calculates the scan of each non-global histogram, saving it in different lines of the vertical scan
__global__ void calculateVerticalScan(int *histograms, int *Vscan, int* Hscan, int segCount, int hist_count) {
    int posiX = threadIdx.x;         // starts as thread ID

    //--

    while(posiX < segCount) {
      // Loop while inside the histogram's segments

      int sum = 0;

      for (int posiY = 0; posiY < hist_count; posiY++) {
        Vscan[posiX + (posiY*segCount)] = sum + Hscan[posiX];
        sum += histograms[posiX + (posiY*segCount)];
      }

      //--

      posiX += blockDim.x;
      // jumps to the next unprocessed column
    }

    //--

    __syncthreads();
}



//---------------------------------------------------------------



// Uses the consultation table to separate the groups of numbers according to their bins
// saves in output device memory
__global__ void PartitionKernel(u_int* output, u_int* input, int* table, int arraySize, int segSize, int segCount, u_int minVal, u_int maxVal, u_int binWidth) {
  int posiX = threadIdx.x;
  int blkDiff = (blockIdx.x * segSize);

  //--

  while((posiX < segSize) && ((posiX+blkDiff) < arraySize)) {
    // while inside the block scope and inside the array

    //                     X                                                                Y
    int tableID = BINFIND(minVal, maxVal, input[posiX+blkDiff], binWidth, segCount) + (blockIdx.x*segCount);
    int posi = atomicAdd(&table[tableID], 1);

    output[posi] = input[posiX+blkDiff];

    // jumps to next unprocessed element
    posiX += blockDim.x;
  }

  //--

  __syncthreads();
}



//---------------------------------------------------------------



int main() {
    ////======= INPUT VECTORS
    u_int *d_input;
    u_int* h_input = genRandomArray(ARRAYSIZE);
    //u_int h_input[] = {2, 4, 33, 27, 8, 10, 42, 3, 12, 21, 10, 12, 15, 27, 38, 45, 18, 22};

    // Allocate memory on the device
    hipMalloc((void**)&d_input, ARRAYSIZE * sizeof(u_int));
    hipMemcpy(d_input, h_input, ARRAYSIZE * sizeof(u_int), hipMemcpyHostToDevice);

    u_int min = getMin(h_input, ARRAYSIZE);
    u_int max = getMax(h_input, ARRAYSIZE);
    u_int binWidth = getBinSize(min, max, HIST_SEGMENTATIONS);

    uintPrintArray(h_input, ARRAYSIZE);
    printSegmentations(min, max, h_input, ARRAYSIZE, HIST_SEGMENTATIONS);

    ////======= HISTOGRAM

    int *d_histograms;          // each block has a histogram
    int *d_histogram_total;     // the sum of all histograms together
    int h_histograms[BLOCKS][HIST_SEGMENTATIONS] = {0};
    int h_histogram_total[HIST_SEGMENTATIONS] = {0};

    // Allocate memory on the device
    hipMalloc((void**)&d_histograms, HISTOGRAM * sizeof(int));
    hipMemset(d_histograms, 0, HISTOGRAM * sizeof(int));  // Initialize histograms to 0

    hipMalloc((void**)&d_histogram_total, HIST_SEGMENTATIONS * sizeof(int));
    hipMemset(d_histograms, 0, HIST_SEGMENTATIONS * sizeof(int));  // Initialize histograms to 0

    ////======= SCAN

    int *d_scan;                                    // the scan of the global histogram
    int *d_verticalScan;                            // the scan of each histogram
    int h_scan[HIST_SEGMENTATIONS] = {0};
    int h_verticalScan[BLOCKS][HIST_SEGMENTATIONS] = {0};

    hipMalloc((void**)&d_scan, HIST_SEGMENTATIONS * sizeof(int));
    hipMemset(d_scan, 0, HIST_SEGMENTATIONS * sizeof(int));  // Initialize histograms to 0

    hipMalloc((void**)&d_verticalScan, HISTOGRAM * sizeof(int));
    hipMemset(d_verticalScan, 0, HISTOGRAM * sizeof(int));  // Initialize histograms to 0

    ////======= PARTITION array

    u_int *d_partition;                                    // the final array memory
    u_int h_partition[ARRAYSIZE] = {0};

    hipMalloc((void**)&d_partition, ARRAYSIZE * sizeof(u_int));
    hipMemset(d_partition, 0, ARRAYSIZE * sizeof(u_int));  // Initialize histograms to 0

    ////=======////======= KERNEL 1 - HIST

    // Launch kernel
    calculateHistogram<<<BLOCKS, THREADS, SEG_SIZE>>>(d_input, d_histograms, d_histogram_total, ARRAYSIZE, SEG_SIZE, HIST_SEGMENTATIONS, min, max, binWidth);

    ////=======////======= KERNEL 2+3 - SCAN

    // Launch kernel horizontal scan
    calculateHorizontalScan<<<1, THREADS>>>(d_histogram_total, d_scan, HIST_SEGMENTATIONS);
    hipMemcpy(h_histogram_total, d_histogram_total, HIST_SEGMENTATIONS * sizeof(int), hipMemcpyDeviceToHost);

    // Launch kernel vertical scan
    calculateVerticalScan<<<1, THREADS>>>(d_histograms, d_verticalScan, d_scan, HIST_SEGMENTATIONS, BLOCKS);

    ////=======////======= KERNEL 4 - VECTOR SUM

    // Launch kernel that uses the information in vector sum to ordenate
    PartitionKernel<<<BLOCKS, THREADS>>>(d_partition, d_input, d_verticalScan, ARRAYSIZE, SEG_SIZE, HIST_SEGMENTATIONS, min, max, binWidth);
    hipMemcpy(h_partition, d_partition, ARRAYSIZE * sizeof(u_int), hipMemcpyDeviceToHost);

    ////=======////======= KERNEL 6 - Bitonic Sort

    // launch kernel that that sorts the inside of each bin partition
    u_int start;
    u_int end = 0;
    for (int bin = 0; bin < HIST_SEGMENTATIONS; bin++) {
      // call a bitonic sort for every bin
      start = end;
      end += h_histogram_total[bin];
      
      //--

      bitonicSortProxy(d_partition, start, end); 
    }

    ////=======////======= COPY BACK

    // Copy result back to host
    hipMemcpy(h_histograms, d_histograms, HISTOGRAM * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(h_scan, d_scan, HIST_SEGMENTATIONS * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(h_verticalScan, d_verticalScan, HISTOGRAM * sizeof(int), hipMemcpyDeviceToHost);

    ////======= PRINT RESULT

    // Print the histograms
    for (int i = 0; i < BLOCKS; i++) {
        std::cout << "Histogram " << i << ": ";
        for (int j = 0; j < HIST_SEGMENTATIONS; j++)
            std::cout << h_histograms[i][j] << " ";
        std::cout << std::endl;
    }

    std::cout << "Histogram_total: ";
    intPrintArray(h_histogram_total, HIST_SEGMENTATIONS);

    std::cout << "Hist Scan: ";
    intPrintArray(h_scan, HIST_SEGMENTATIONS);

    // Print the histograms
    for (int i = 0; i < BLOCKS; i++) {
        std::cout << "Hist Vertical Scan " << i << ": ";
        for (int j = 0; j < HIST_SEGMENTATIONS; j++)
            std::cout << h_verticalScan[i][j] << " ";
        std::cout << std::endl;
    }

    uintPrintArray(h_partition, ARRAYSIZE);

    ////======= VERIFY
    if (verifySort(h_partition, ARRAYSIZE)) {
      printf("\n\nARRAY CORRETO!!!\n\n");
    } else {
      printf("\n\nMerda!!!\n\n");
    }

    ////======= FREE MEMORY

    // Free device memory
    hipFree(d_input);
    hipFree(d_histograms);
    hipFree(d_histogram_total);
    hipFree(d_scan);
    hipFree(d_verticalScan);

    return 0;
}

