#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include "simpleSort.cuh"
#include "histogramM.cuh"

#define BINSTART(min, binSize, i) ((u_int)((binWidth*i)+min))
#define BINEND(min, binSize, i) ((u_int)(BINSTART(min, binSize, (i+1))-1))
#define BINFIND(min, max, val, binSize, binQtd) (val >= max ? binQtd-1 : (val - min) / binSize)



//--------------------------------------------------------------------------



// returns the size of the number group of each bin
// needs some strange calculations due to precision error
u_int H_getBinSize(u_int min, u_int max, int segCount) {
  u_int binSize = max - min;
  if ((binSize % segCount) == 0) {
    // complete division
    binSize /= segCount;
  } else {
    // incomplete division
    binSize /= segCount;
    binSize++;
  }

  return binSize;
}



//---------------------------------------------------------------------------------



// return the min of a device array
__global__ u_int H_getMin(u_int* d_array, u_int size) {
    __shared__ int min;

    u_int posi = (blockIdx.x*blockDim.x) + threadIdx.x;

    while(posi < size) {

        min = atomicMin(&d_array[posi], min);

        posi += blockDim.x;
    }

    __syncthreads();

    //--

    return min;
}


// return the max of a device array
__global__ u_int H_getMax(u_int* d_array, u_int size) {
    __shared__ int max;

    u_int posi = (blockIdx.x*blockDim.x) + threadIdx.x;

    while(posi < size) {

        max = atomicMax(&d_array[posi], max);

        posi += blockDim.x;
    }

    __syncthreads();

    //--

    return max;
}



//--------------------------------------------------------------------------


// Kernel para calcular histogramas em particoes
// Cada bloco eh responsavel por um histograma (linha da matriz)
__global__ void H_getHistogram(u_int *HH, u_int *Hg, u_int h, u_int *Input, u_int nElements, u_int nMin, u_int nMax, u_int segSize, u_int binWidth) {
    // Alloca shared memory para UM histograma
    extern __shared__ int _HH[];
    if (threadIdx.x < h) { _HH[threadIdx.x] = 0; }
    __syncthreads();

    //---

    // Inicio da particao no vetor
    int blcStart = (blockIdx.x * segSize);    // bloco positionado na frente daquele que veio anterior a ele
    int thrdPosi = threadIdx.x;              // 1 elemento por thread, starts as exactly the thread.x

    while(thrdPosi < segSize && ((blcStart+thrdPosi) < nElements)) {
        // Loop enquanto a thread estiver resolvendo elementos validos dentro do bloco e do array
        u_int val = Input[blcStart + thrdPosi];    // get value
        int posi = BINFIND(nMin, nMax, val, binWidth, h);
        atomicAdd(&_HH[posi], 1);  // add to its corresponding segment
        atomicAdd(&Hg[posi], 1);  // add to its corresponding segment

        thrdPosi += blockDim.x; // thread pula para frente, garantindo que nao ira processar um valor ja processado
    }

    __syncthreads();

    //--

    // Passa os resultados da shared memory para matriz
    // deixar isso a cargo da thread 0 eh mais modular que mandar uma pra uma
    if (threadIdx.x < h)
      atomicAdd(&HH[(blockIdx.x * h) + threadIdx.x], _HH[threadIdx.x]);
    __syncthreads();
}


// calculates the scan of the global histogram and saves it into the horizontal scan
__global__ void H_horizontalScan(u_int *Hg, u_int *SHg, u_int h) {
    // Obtem shared memory para o histogram horizontal
    extern __shared__ u_int _SHg[];
    if (threadIdx.x < h) { _SHg[threadIdx.x] = 0; }
    __syncthreads();

    //--

    u_int thrdPosi = threadIdx.x;         // starts as thread ID

    //--

    while (thrdPosi < h) {
      // Loop while inside the histogram
      u_int sum = 0;
      for (int i = thrdPosi-1; i >= 0; i--) {
        sum += Hg[i]; // makes the individual sum of every index before this one
      }
      _SHg[thrdPosi] = sum;

      //--

      thrdPosi += blockDim.x; // go to the next element
    }

    __syncthreads();

    //--

    // Passa os resultados da shared memory para o scan
    // deixar isso a cargo da thread 0 eh mais modular que mandar uma pra uma
    if (threadIdx.x < h)
      SHg[threadIdx.x] = _SHg[threadIdx.x];
    __syncthreads();
}


// calculates the scan of each non-global histogram, saving it in different lines of the vertical scan
__global__ void H_verticalScan(u_int *HH, u_int *PSv, u_int h) {
    // Obtem shared memory para o histogram horizontal
    extern __shared__ u_int _PSv[];
    if (threadIdx.x < h) { _PSv[threadIdx.x] = 0; }
    __syncthreads();

    //--

    u_int thrdPosi = threadIdx.x;     // Thread por coluna

    //--

    while (thrdPosi < h) {
      int sum = 0;
      for (int i=blockIdx.x-1; i>=0; i--){
        sum += HH[i*h + thrdPosi];
      }
      _PSv[thrdPosi] = sum;

      thrdPosi += blockDim.x; // go to the next element
    }
    __syncthreads();

    //--

    // Passa os resultados da shared memory para o scan
    // deixar isso a cargo da thread 0 eh mais modular que mandar uma pra uma
    if (threadIdx.x < h)
      PSv[blockIdx.x*h + threadIdx.x] = _PSv[threadIdx.x];
    __syncthreads();
}



//--------------------------------------------------------------------------



// Uses the consultation table to separate the groups of numbers according to their bins
// saves in output device memory
__global__ void H_Partitioner(u_int *HH, u_int *SHg, u_int *PSv, u_int h, u_int *Input, u_int *Output, u_int nElements, u_int nMin, u_int nMax, u_int segSize, u_int binWidth) {
    extern __shared__ u_int _HLsh[];
    if (threadIdx.x < h) { _HLsh[threadIdx.x] = 0; }
    __syncthreads();

    // Thread ID and total threads
    u_int thrdPosi = threadIdx.x; 
    u_int totalThreads = blockDim.x;

    // Calculate the indices for shared memory
    while (thrdPosi < h) {
        _HLsh[thrdPosi] = SHg[thrdPosi] + PSv[blockIdx.x * h + thrdPosi];
        thrdPosi += totalThreads;
    }
    __syncthreads();

    // Reset thread position for the next phase
    thrdPosi = threadIdx.x;

    // Process elements in the segment
    while (thrdPosi < segSize && ((blockIdx.x * segSize + thrdPosi) < nElements)) {
        u_int val = Input[blockIdx.x * segSize + thrdPosi]; 
        u_int posi = BINFIND(nMin, nMax, val, binWidth, h);

        // Atomic operation to update the output array
        u_int index = atomicAdd(&_HLsh[posi], 1);    // Get the current position and increment it atomically
        if (index < nElements)
          Output[index] = val;                         // Write the value to the output array

        thrdPosi += totalThreads;
    }
    __syncthreads();
}